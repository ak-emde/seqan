#include "hip/hip_runtime.h"
// ==========================================================================
//                                cuda_mapper
// ==========================================================================
// Copyright (c) 2006-2013, Knut Reinert, FU Berlin
// All rights reserved.
//
// Redistribution and use in source and binary forms, with or without
// modification, are permitted provided that the following conditions are met:
//
//     * Redistributions of source code must retain the above copyright
//       notice, this list of conditions and the following disclaimer.
//     * Redistributions in binary form must reproduce the above copyright
//       notice, this list of conditions and the following disclaimer in the
//       documentation and/or other materials provided with the distribution.
//     * Neither the name of Knut Reinert or the FU Berlin nor the names of
//       its contributors may be used to endorse or promote products derived
//       from this software without specific prior written permission.
//
// THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
// AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
// IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE
// ARE DISCLAIMED. IN NO EVENT SHALL KNUT REINERT OR THE FU BERLIN BE LIABLE
// FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
// DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
// SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
// CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT
// LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY
// OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH
// DAMAGE.
//
// ==========================================================================
// Author: Enrico Siragusa <enrico.siragusa@fu-berlin.de>
// ==========================================================================

// ============================================================================
// Prerequisites
// ============================================================================

#include "kernels.h"

using namespace seqan;

// --------------------------------------------------------------------------
// Function mapReadsGPU()
// --------------------------------------------------------------------------

template <typename TIndexView, typename TReadSeqsView>
__global__ void
mapReadsGPU(TIndexView index, TReadSeqsView readSeqs)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

//    mapRead(index, readSeqs[idx]);

    typename Iterator<TIndexView, TopDown<> >::Type it(index);

    unsigned occurrences = goDown(it, readSeqs[idx]) ? countOccurrences(it) : 0;

    printf("index=%i, occurrences=%d\n", idx, occurrences);
}

// --------------------------------------------------------------------------
// Function mapReads()                                                  [GPU]
// --------------------------------------------------------------------------

void
mapReads(Index<StringSet<String<Dna>, Owner<ConcatDirect<> > >, FMIndex<> > & index,
         StringSet<String<Dna>, Owner<ConcatDirect<> > > & readSeqs,
         GPU const & /* tag */)
{
    typedef Index<StringSet<String<Dna>, Owner<ConcatDirect<> > >, FMIndex<> >  TIndex;
    typedef StringSet<String<Dna>, Owner<ConcatDirect<> > >                     TReadSeqs;

    typedef typename Device<TIndex>::Type               TDeviceIndex;
    typedef typename Device<TReadSeqs>::Type            TDeviceReadSeqs;

    // Copy index to device.
    TDeviceIndex deviceIndex;
    assign(deviceIndex, index);

    // Copy read seqs to device.
    TDeviceReadSeqs deviceReadSeqs;
    assign(deviceReadSeqs, readSeqs);

    // Launch kernel.
    mapReadsGPU<<<10,100>>>(view(deviceIndex), view(deviceReadSeqs));
    hipDeviceSynchronize();
}
